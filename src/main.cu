
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <hipblas.h>

constexpr unsigned max_log_N = 15;
constexpr unsigned min_log_N = 7;

constexpr unsigned test_count = 1u << 7;

int main() {
	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	double *A_ptr, *x_ptr, *y_ptr;
	const auto max_N = 1lu << max_log_N;
	hipMalloc(&A_ptr, sizeof(double) * max_N * max_N);
	hipMemset(A_ptr, 0, sizeof(double) * max_N * max_N);
	hipMalloc(&x_ptr, sizeof(double) * max_N);
	hipMemset(x_ptr, 0, sizeof(double) * max_N);
	hipMalloc(&y_ptr, sizeof(double) * max_N);
	hipMemset(y_ptr, 0, sizeof(double) * max_N);

	std::printf("N,bandwidth_in_tbyteps,throughput_in_tflops\n");

	const double alpha = 1., beta = 0.;
	for (unsigned log_N = min_log_N; log_N <= max_log_N; log_N++) {
		const auto N = 1lu << log_N;
		hipblasDgemv(cublas_handle, HIPBLAS_OP_N, N, N, &alpha, A_ptr, N, x_ptr, 1, &beta, y_ptr, 1);

		hipDeviceSynchronize();
		const auto start_clock = std::chrono::system_clock::now();

		for (unsigned i = 0; i < test_count; i++ ) {
			hipblasDgemv(cublas_handle, HIPBLAS_OP_N, N, N, &alpha, A_ptr, N, x_ptr, 1, &beta, y_ptr, 1);
		}

		hipDeviceSynchronize();
		const auto end_clock = std::chrono::system_clock::now();
		const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6 / test_count;

		std::printf("%lu,%e,%e\n",
				N,
				(N * N + 2 * N) * sizeof(double) / elapsed_time * 1e-12,
				(2 * N * N) / elapsed_time * 1e-12
				);
	}

	hipFree(A_ptr);
	hipFree(x_ptr);
	hipFree(y_ptr);
}
